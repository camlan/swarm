#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "Windows.h"

__global__ void setupKernel(hiprandState* state, uint64_t seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void leaveFermone(double* fermoneMap1D, int* citySequences, double * distances, double fermoneIncrease, int cityCount) {
    int i = blockIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    double leftFermone = fermoneIncrease / distances[i];
    if (j > 0) {
        fermoneMap1D[citySequences[j - 1] * cityCount + citySequences[j]] += leftFermone;
        fermoneMap1D[citySequences[j] * cityCount + citySequences[j-1]] += leftFermone;
    }

    if (j == 0) {
        fermoneMap1D[citySequences[i * cityCount] * cityCount + citySequences[i * cityCount + cityCount - 1]] += leftFermone;
        fermoneMap1D[citySequences[i * cityCount + cityCount - 1] * cityCount + citySequences[i * cityCount]] += leftFermone;
    }

}

__global__ void scaleMapKernel(double* scaledMap, double* baseMap, double scaler, unsigned int size) {
    int i = threadIdx.x;
    if (i < size) {
        if (baseMap[i] > 0) {
            scaledMap[i] = scaler / baseMap[i];
        }
        else {
            scaledMap[i] = 0;
        }
    }
}

__global__ void evaporateFermoneKernel(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
    int i = threadIdx.x; 
    if (i < size) {
        fermoneMap1D[i] *= fermoneEvaporation;
    }
}

__device__ double calculatePathSelectionProbalitity(double distance, double distanceImportance, double fermone, double fermoneImportance) {
    return pow(distance, distanceImportance) * pow(fermone, fermoneImportance);
}

__device__ void calculatePathsSelectionProbabilies(double* cityMap1D, double* fermoneMap1D, double fermoneImportance, double distanceImportance, int currentCity, char* visited, int citiesCount, int * nextCitiesIndex, double * nextCityProbability) {

    double* distances = (cityMap1D + (citiesCount * currentCity));
    double* fermones = (fermoneMap1D + (citiesCount * currentCity));
    double totalProbabilty = 0;

    for (int i = 0; i < citiesCount; i++) {
        nextCityProbability[i] = -1000;
        nextCitiesIndex[i]= -1;
    }

    int position = 0;
    for (int i = 0; i < citiesCount; i++) {
        if (distances[i] > 0 && !visited[i]) {
            nextCitiesIndex[position] = i;
            nextCityProbability[position] = calculatePathSelectionProbalitity(distances[i], distanceImportance, fermones[i], fermoneImportance);
            totalProbabilty += nextCityProbability[position];
 
            position++;
        }
    }

    
    nextCityProbability[0] /= totalProbabilty;

    for (int i = 1; i < position; i++) {
        nextCityProbability[i] /= totalProbabilty;
        nextCityProbability[i] += nextCityProbability[i - 1];
    }

}

__device__ int selectNexyCity(int* nextCitiesIndex, double* nextCityProbability, double randomSelector, int citiesCount) {

    int pos = 0;
    for (int i = 0; i < citiesCount; i++) {
        if (nextCitiesIndex[i] < 0) {
            break;
        }

        if (randomSelector < nextCityProbability[i]) {
            return nextCitiesIndex[i];
        }
        pos++;
    }
    return nextCitiesIndex[pos-1];
}

__device__ double calculatePathDistance(int* citySequence, unsigned int citiesCount, double* distanceMap1D) {
    double distance = 0;

    for (int i = 1; i < citiesCount; i++) {
        distance += *(distanceMap1D + (*(citySequence + i - 1) * citiesCount) + *(citySequence + i));
    }
    distance += *(distanceMap1D + (*(citySequence + citiesCount - 1) * citiesCount) + *(citySequence));

    return distance;
}

__global__ void moveAnt(double* cityMap1D, double* distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences, hiprandState* state, char * visited, int * citySequence, int * nextCityProbabilitiesIndex, double * nextCityProbabilitiesProbability) {
    int ant = threadIdx.x; 


    visited = (visited + ant * citiesCount);
    citySequence = (citySequence + ant * citiesCount);
    nextCityProbabilitiesIndex = (nextCityProbabilitiesIndex + ant * citiesCount);
    nextCityProbabilitiesProbability = (nextCityProbabilitiesProbability + ant * citiesCount);


    for (int i = 0; i < citiesCount; i++) {
        visited[i] = 0;
        citySequence[i] = 0;
        nextCityProbabilitiesIndex[i] = 0;
        nextCityProbabilitiesProbability[i] = 0;
    }


    int currentCity = ant;
    hiprandState localState = state[ant];
    double r;
    for (int i = 0; i < citiesCount; i++) {
       visited[currentCity] = 1;
       citySequence[i] = currentCity;
       calculatePathsSelectionProbabilies(cityMap1D, fermoneMap1D, fermoneImportance, distanceImportance, currentCity, visited, citiesCount, nextCityProbabilitiesIndex, nextCityProbabilitiesProbability);
       r = hiprand_uniform(&localState);
       currentCity = selectNexyCity(nextCityProbabilitiesIndex, nextCityProbabilitiesProbability, r, citiesCount);
    }
    
    distances[ant] = calculatePathDistance(citySequence, citiesCount, distanceMap1D);

    for (int i = 0; i < citiesCount; i++) {
        citySequences[ant * citiesCount + i] = citySequence[i];
    }
}



void evaporateFermone(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
    double* dev_fermone_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_fermone_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    evaporateFermoneKernel << <1, size >> > (dev_fermone_map, size, fermoneEvaporation);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(fermoneMap1D, dev_fermone_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    Error:
        hipFree(dev_fermone_map);
}

void scaleMap(double* scaledMap, const double* baseMap, double scaler, unsigned int size) {
    double* dev_scaled_map = 0;
    double* dev_base_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_scaled_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_base_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_base_map, baseMap, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    scaleMapKernel << <1, size >> > (dev_scaled_map, dev_base_map, scaler, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(scaledMap, dev_scaled_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    Error:
        hipFree(dev_scaled_map);
        hipFree(dev_base_map);
}

void moveAnts(double* cityMap1D, double* distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences) {
    double* dev_city_map = 0;
    double* dev_distance_map = 0;
    double* dev_fermone_map = 0;
    double* dev_distances = 0;
    int* dev_city_sequences = 0;
    hipError_t cudaStatus;

    hiprandState* devStates=0;


    char * dev_visited;
    int * dev_citySequence;
    int * dev_nextCityProbabilitiesIndex;
    double *  dev_nextCityProbabilitiesProbability;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_visited, citiesCount * citiesCount * sizeof(char));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_citySequence, citiesCount * citiesCount * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_nextCityProbabilitiesIndex, citiesCount * citiesCount * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nextCityProbabilitiesProbability, citiesCount * citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }



    cudaStatus = hipMalloc((void**)&dev_city_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distance_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_fermone_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distances, citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_city_sequences, mapSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&devStates, citiesCount * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_city_map, cityMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_distance_map, distanceMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }
    

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    unsigned __int64 startTime;
    QueryPerformanceCounter((LARGE_INTEGER*)&startTime);

    setupKernel << <1, citiesCount >> > (devStates, startTime);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    moveAnt << <1, citiesCount >> > (dev_city_map, dev_distance_map, dev_fermone_map, mapSize, citiesCount, fermoneImportance, distanceImportance, dev_distances, dev_city_sequences, devStates, dev_visited, dev_citySequence, dev_nextCityProbabilitiesIndex, dev_nextCityProbabilitiesProbability);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(distances, dev_distances, citiesCount * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(citySequences, dev_city_sequences, mapSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(dev_city_map);
    hipFree(dev_distance_map);
    hipFree(dev_fermone_map);
    hipFree(dev_distances);
    hipFree(dev_city_sequences);

    hipFree(dev_visited);
    hipFree(dev_citySequence);
    hipFree(dev_nextCityProbabilitiesIndex);
    hipFree(dev_nextCityProbabilitiesProbability);
}

void leaveFermones(double* fermoneMap1D, int* citySequences, double* distances, double fermoneIncrease, unsigned int mapSize, unsigned int citiesCount) {

    double* dev_fermone_map = 0;
    int* dev_city_sequences = 0;
    double* dev_distancecs = 0;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_fermone_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_city_sequences, mapSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distancecs, citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_city_sequences, citySequences, mapSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_distancecs, distances, citiesCount * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    leaveFermone << <citiesCount, citiesCount >> > (dev_fermone_map, dev_city_sequences, dev_distancecs, fermoneIncrease, citiesCount);


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(fermoneMap1D, dev_fermone_map, mapSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


Error:
    hipFree(dev_fermone_map);
    hipFree(dev_city_sequences);
    hipFree(dev_distancecs);
}

extern "C" {

    void scale_city_matrix_wrp(double* flatScaledCityMap, const double* flatCityMap, unsigned int size, double distanceScaler) {
        scaleMap(flatScaledCityMap, flatCityMap, distanceScaler, size);
    }

    void evaporate_fermone_wrp(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
        evaporateFermone(fermoneMap1D, size, fermoneEvaporation);
    }


    void move_ants_wrp(double* cityMap1D, double * distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences) {
        moveAnts(cityMap1D, distanceMap1D, fermoneMap1D, mapSize, citiesCount, fermoneImportance, distanceImportance, distances, citySequences);
    }

    void leave_fermone_wrp(double* fermoneMap1D, int* citySequences, double * distances, double fermoneIncrease, unsigned int mapSize, unsigned int cityCount) {
        leaveFermones(fermoneMap1D, citySequences, distances, fermoneIncrease, mapSize, cityCount);
    }
}
