#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <stdlib.h>
#include <math.h>
#include <time.h>
//#include <stdio.h>

struct NextCity {
    int cityIndex;
    double probability;
};

__global__ void setupKernel(hiprandState* state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void leaveFermone(double* fermoneMap1D, int* citySequences, double * distances, double fermoneIncrease, int cityCount) {
    int i = blockIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    double leftFermone = fermoneIncrease / distances[i];
    if (j > 0) {
        fermoneMap1D[citySequences[j - 1] * cityCount + citySequences[j]] += leftFermone;
        fermoneMap1D[citySequences[j] * cityCount + citySequences[j-1]] += leftFermone;
    }

    if (j == 0) {
        fermoneMap1D[citySequences[i * cityCount] * cityCount + citySequences[i * cityCount + cityCount - 1]] += leftFermone;
        fermoneMap1D[citySequences[i * cityCount + cityCount - 1] * cityCount + citySequences[i * cityCount]] += leftFermone;
    }

}

__global__ void scaleMapKernel(double* scaledMap, double* baseMap, double scaler, unsigned int size) {
    int i = threadIdx.x;
    if (i < size) {
        if (baseMap[i] > 0) {
            scaledMap[i] = scaler / baseMap[i];
        }
        else {
            scaledMap[i] = 0;
        }
    }
}

__global__ void evaporateFermoneKernel(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
    int i = threadIdx.x; 
    if (i < size) {
        fermoneMap1D[i] *= fermoneEvaporation;
    }
}

__device__ double calculatePathSelectionProbalitity(double distance, double distanceImportance, double fermone, double fermoneImportance) {
    return pow(distance, distanceImportance) * pow(fermone, fermoneImportance);
}

__device__ void calculatePathsSelectionProbabilies(double* cityMap1D, double* fermoneMap1D, double fermoneImportance, double distanceImportance, int currentCity, char* visited, int citiesCount, int * nextCitiesIndex, double * nextCityProbability) {
   //  NextCity* nc;

    double* distances = (cityMap1D + (citiesCount * currentCity));
    double* fermones = (fermoneMap1D + (citiesCount * currentCity));
    double totalProbabilty = 0;
    //nc = (NextCity*)malloc(citiesCount * sizeof(struct NextCity));

    for (int i = 0; i < citiesCount; i++) {
        nextCityProbability[i] = -1000;
        nextCitiesIndex[i]= -1;
    }

    int position = 0;
    for (int i = 0; i < citiesCount; i++) {
        if (distances[i] > 0 && !visited[i]) {
            nextCitiesIndex[position] = i;
            nextCityProbability[position] = calculatePathSelectionProbalitity(distances[i], distanceImportance, fermones[i], fermoneImportance);
            totalProbabilty += nextCityProbability[position];
            //nc->next = (NextCity*)malloc(sizeof(struct NextCity));
            //nc = nc->next;
            //nc->next = 0;
            //nc->probability = -100;
            position++;
        }
    }

    
    nextCityProbability[0] /= totalProbabilty;

    for (int i = 1; i < position; i++) {
        nextCityProbability[i] /= totalProbabilty;
        nextCityProbability[i] += nextCityProbability[i - 1];
    }

    //while (nc->next && nc->next->probability > 0) {
    //    nc->next->probability /= totalProbabilty;
    //    nc->next->probability += nc->probability;
    //    nc = nc->next;
    //}

    //return nc;
}

__device__ int selectNexyCity(int* nextCitiesIndex, double* nextCityProbability, double randomSelector, int citiesCount) {
    //for (int i = 0; i < citiesCount; i++) {
    //    if (randomSelector < nc[i].probability) {

    //    }
    //}


    // return nc[0].cityIndex;
    int pos = 0;
    for (int i = 0; i < citiesCount; i++) {
        if (nextCitiesIndex[i] < 0) {
            break;
        }

        if (randomSelector < nextCityProbability[i]) {
            return nextCitiesIndex[i];
        }
        pos++;
    }
    return nextCitiesIndex[pos-1];

    //while (nc) {
    //    if (randomSelector < nc->probability) {
    //        return nc->cityIndex;
    //    }
    //    if (!nc->next) {
    //        return nc->cityIndex;
    //    }
    //    nc = nc->next;
    //}
    //return nc->cityIndex;
}

__device__ double calculatePathDistance(int* citySequence, unsigned int citiesCount, double* distanceMap1D) {
    double distance = 0;

    for (int i = 1; i < citiesCount; i++) {
        distance += *(distanceMap1D + (*(citySequence + i - 1) * citiesCount) + *(citySequence + i));
    }
    distance += *(distanceMap1D + (*(citySequence + citiesCount - 1) * citiesCount) + *(citySequence));

    return distance;
}

__global__ void moveAnt(double* cityMap1D, double* distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences, hiprandState* state) {
    int ant = threadIdx.x; // TODO consider different block/thread structure
    char visited[4];
    int citySequence[4];
    // NextCity nextCityProbabilities[4]; //= (NextCity**)malloc(sizeof(struct NextCity));
    int nextCityProbabilitiesIndex[4];
    double nextCityProbabilitiesProbability[4];


    int currentCity = ant;
    hiprandState localState = state[ant];
    double r;
    for (int i = 0; i < citiesCount; i++) {
        visited[currentCity] = 1;
        citySequence[i] = currentCity;
        calculatePathsSelectionProbabilies(cityMap1D, fermoneMap1D, fermoneImportance, distanceImportance, currentCity, visited, citiesCount, nextCityProbabilitiesIndex, nextCityProbabilitiesProbability);
        r = hiprand_uniform(&localState);
        currentCity = selectNexyCity(nextCityProbabilitiesIndex, nextCityProbabilitiesProbability, r, citiesCount);
    }

    distances[ant] = calculatePathDistance(citySequence, citiesCount, distanceMap1D);

    for (int i = 0; i < citiesCount; i++) {
        citySequences[ant * citiesCount + i] = citySequence[i];
    }

    //NextCity* tmp = nextCityProbabilities;

    //int i = 0;
    //while (tmp && i < citiesCount) {
    //    free(tmp);
    //    tmp = (nextCityProbabilities + ++i);
    //}

    //while (nextCityProbabilities->next) {
    //    tmp = nextCityProbabilities->next;
    //    free(nextCityProbabilities);
    //    nextCityProbabilities = tmp;
    //}

    // free(nextCityProbabilities);
}

//__global__ void moveAnt_old(double* cityMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int *citySequences, hiprandState* state) {
//    int ant = threadIdx.x; // TODO consider different block/thread structure
//    char visited [100]; // TODO figure out proper memory allocation technique within device to share accross all threads (should be citiesCount)
//    int citySequence[100]; // TODO figure out proper memory allocation technique within device to share accross all threads (should be citiesCount)
//    NextCity* nextCityProbabilities=0;
//
//    int currentCity = ant;
//    hiprandState localState = state[ant];
//    for (int i = 0; i < citiesCount; i++) {
//        visited[currentCity] = 1;
//        citySequence[i] = currentCity;
//        nextCityProbabilities = calculatePathsSelectionProbabilies(cityMap1D, fermoneMap1D, fermoneImportance, distanceImportance, currentCity, visited, citiesCount, nextCityProbabilities);
//        double r = (double)hiprand_uniform(&localState);
//        currentCity = selectNexyCity(nextCityProbabilities, r, citiesCount);
//    }
//    double distance = calculatePathDistance(citySequence, citiesCount, cityMap1D);
//
//    distances[ant] = distance;
//    
//    //TODO transer for loop to parallel ??
//    for (int i = 0; i < citiesCount; i++) {
//        citySequences[ant * citiesCount + i] = citySequence[i];
//    }
//}

void evaporateFermone(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
    double* dev_fermone_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_fermone_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    evaporateFermoneKernel << <1, size >> > (dev_fermone_map, size, fermoneEvaporation);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(fermoneMap1D, dev_fermone_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    Error:
        hipFree(dev_fermone_map);
}

void scaleMap(double* scaledMap, const double* baseMap, double scaler, unsigned int size) {
    double* dev_scaled_map = 0;
    double* dev_base_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_scaled_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_base_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_base_map, baseMap, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    scaleMapKernel << <1, size >> > (dev_scaled_map, dev_base_map, scaler, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(scaledMap, dev_scaled_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    Error:
        hipFree(dev_scaled_map);
        hipFree(dev_base_map);
}

void moveAnts(double* cityMap1D, double* distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences) {
    double* dev_city_map = 0;
    double* dev_distance_map = 0;
    double* dev_fermone_map = 0;
    double* dev_distances = 0;
    int* dev_city_sequences = 0;
    hipError_t cudaStatus;

    hiprandState* devStates=0;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_city_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distance_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_fermone_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distances, citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_city_sequences, mapSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&devStates, citiesCount * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_city_map, cityMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_distance_map, distanceMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }
    

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    setupKernel << <1, citiesCount >> > (devStates);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    moveAnt << <1, citiesCount >> > (dev_city_map, dev_distance_map, dev_fermone_map, mapSize, citiesCount, fermoneImportance, distanceImportance, dev_distances, dev_city_sequences, devStates);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(distances, dev_distances, citiesCount * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(citySequences, dev_city_sequences, mapSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(dev_city_map);
    hipFree(dev_distance_map);
    hipFree(dev_fermone_map);
    hipFree(dev_distances);
    hipFree(dev_city_sequences);
}

void leaveFermones(double* fermoneMap1D, int* citySequences, double* distances, double fermoneIncrease, unsigned int mapSize, unsigned int citiesCount) {

    double* dev_fermone_map = 0;
    int* dev_city_sequences = 0;
    double* dev_distancecs = 0;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_fermone_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_city_sequences, mapSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distancecs, citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_city_sequences, citySequences, mapSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_distancecs, distances, citiesCount * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    leaveFermone << <citiesCount, citiesCount >> > (dev_fermone_map, dev_city_sequences, dev_distancecs, fermoneIncrease, citiesCount);


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(fermoneMap1D, dev_fermone_map, mapSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


Error:
    hipFree(dev_fermone_map);
    hipFree(dev_city_sequences);
    hipFree(dev_distancecs);
}

extern "C" {

    void scale_city_matrix_wrp(double* flatScaledCityMap, const double* flatCityMap, unsigned int size, double distanceScaler) {
        scaleMap(flatScaledCityMap, flatCityMap, distanceScaler, size);
    }

    void evaporate_fermone_wrp(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
        evaporateFermone(fermoneMap1D, size, fermoneEvaporation);
    }


    void move_ants_wrp(double* cityMap1D, double * distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences) {
        moveAnts(cityMap1D, distanceMap1D, fermoneMap1D, mapSize, citiesCount, fermoneImportance, distanceImportance, distances, citySequences);
    }

    void leave_fermone_wrp(double* fermoneMap1D, int* citySequences, double * distances, double fermoneIncrease, unsigned int mapSize, unsigned int cityCount) {
        leaveFermones(fermoneMap1D, citySequences, distances, fermoneIncrease, mapSize, cityCount);
    }
}
