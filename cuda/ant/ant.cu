#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i] * 2;
}

__global__ void scaleMapKernel(double* scaledMap, double* baseMap, double scaler, unsigned int size) {
    int i = threadIdx.x;
    if (i < size) {
        if (baseMap[i] > 0) {
            scaledMap[i] = scaler / baseMap[i];
        }
        else {
            scaledMap[i] = 0;
        }
    }
}

void scaleMap(double* scaledMap, const double* baseMap, double scaler, unsigned int size) {
    double* dev_scaled_map = 0;
    double* dev_base_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_scaled_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_base_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_base_map, baseMap, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    scaleMapKernel << <1, size >> > (dev_scaled_map, dev_base_map, scaler, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(scaledMap, dev_scaled_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    Error:
        hipFree(dev_scaled_map);
        hipFree(dev_base_map);
}

// Helper function for using CUDA to add vectors in parallel.
void addcuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

extern "C" {
    void add_ant_wpr(int* c, const int* a, const int* b, int size) {
        addcuda(c, a, b, size);
    }

    void scale_city_matrix_wrp(double* flatScaledCityMap, const double* flatCityMap, unsigned int size, double distanceScaler) {
        scaleMap(flatScaledCityMap, flatCityMap, distanceScaler, size);
    }
}