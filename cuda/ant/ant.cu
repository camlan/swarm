#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <stdlib.h>
#include <math.h>
#include <time.h>

struct NextCity {
    int cityIndex;
    double probability;
    NextCity* next;
};

__global__ void setupKernel(hiprandState* state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void leaveFermone(double* fermoneMap1D, int* citySequences, double * distances, double fermoneIncrease, int cityCount) {
    int i = blockIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    double leftFermone = fermoneIncrease / distances[i];
    if (j > 0) {
        fermoneMap1D[citySequences[j - 1] * cityCount + citySequences[j]] += leftFermone;
        fermoneMap1D[citySequences[j] * cityCount + citySequences[j-1]] += leftFermone;
    }

    if (j == 0) {
        fermoneMap1D[citySequences[i * cityCount] * cityCount + citySequences[i * cityCount + cityCount - 1]] += leftFermone;
        fermoneMap1D[citySequences[i * cityCount + cityCount - 1] * cityCount + citySequences[i * cityCount]] += leftFermone;
    }

}


__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i] * 2;
}

__global__ void scaleMapKernel(double* scaledMap, double* baseMap, double scaler, unsigned int size) {
    int i = threadIdx.x;
    if (i < size) {
        if (baseMap[i] > 0) {
            scaledMap[i] = scaler / baseMap[i];
        }
        else {
            scaledMap[i] = 0;
        }
    }
}


__global__ void evaporateFermoneKernel(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
    int i = threadIdx.x; 
    if (i < size) {
        fermoneMap1D[i] *= fermoneEvaporation;
    }
}

__device__ double calculatePathSelectionProbalitity(double distance, double distanceImportance, double fermone, double fermoneImportance) {
    return pow(distance, distanceImportance) * pow(fermone, fermoneImportance);
}

__device__ NextCity* calculatePathsSelectionProbabilies(double* cityMap1D, double* fermoneMap1D, double fermoneImportance, double distanceImportance, int currentCity, char* visited, int citiesCount) {
    NextCity* firstCity;
    NextCity* nc;

    double* distances = (cityMap1D + (citiesCount * currentCity));
    double* fermones = (fermoneMap1D + (citiesCount * currentCity));
    double totalProbabilty = 0;
    nc = (NextCity*)malloc(sizeof(struct NextCity));
    firstCity = nc;

    for (int i = 0; i < citiesCount; i++) {
        if (distances[i] > 0 && !visited[i]) {
            nc->cityIndex = i;
            nc->probability = calculatePathSelectionProbalitity(distances[i], distanceImportance, fermones[i], fermoneImportance);
            totalProbabilty += nc->probability;
            nc->next = (NextCity*)malloc(sizeof(struct NextCity));
            nc = nc->next;
            nc->next = 0;
            nc->probability = -100;
        }
    }

    nc = firstCity;
    nc->probability /= totalProbabilty;

    while (nc->next && nc->next->probability > 0) {
        nc->next->probability /= totalProbabilty;
        nc->next->probability += nc->probability;
        nc = nc->next;
    }

    return firstCity;
}

__device__ int selectNexyCity(NextCity* nc, double randomSelector) {
    while (nc) {
        if (randomSelector < nc->probability) {
            return nc->cityIndex;
        }
        if (!nc->next) {
            return nc->cityIndex;
        }
        nc = nc->next;
    }
    return nc->cityIndex;
}

__device__ double calculatePathDistance(int* citySequence, unsigned int citiesCount, double* distanceMap1D) {
    double distance = 0;

    for (int i = 1; i < citiesCount; i++) {
        distance += *(distanceMap1D + (*(citySequence + i - 1) * citiesCount) + *(citySequence + i));
    }
    //TODO add last return path from last to beginning
    return distance;
}

__global__ void moveAnt(double* cityMap1D, double* distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences, hiprandState* state) {
    int ant = threadIdx.x; // TODO consider different block/thread structure
    char visited[20];
    int citySequence[20];
    NextCity* nextCityProbabilities; 

    int currentCity = ant;
    hiprandState localState = state[ant];
    for (int i = 0; i < citiesCount; i++) {
        visited[currentCity] = 1;
        citySequence[i] = currentCity;
        nextCityProbabilities = calculatePathsSelectionProbabilies(cityMap1D, fermoneMap1D, fermoneImportance, distanceImportance, currentCity, visited, citiesCount);
        double r = hiprand_uniform(&localState);
        currentCity = selectNexyCity(nextCityProbabilities, r);
    }

    distances[ant] = calculatePathDistance(citySequence, citiesCount, distanceMap1D);

    for (int i = 0; i < citiesCount; i++) {
        citySequences[ant * citiesCount + i] = citySequence[i];
    }
}

__global__ void moveAnt_old(double* cityMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int *citySequences, hiprandState* state) {
    int ant = threadIdx.x; // TODO consider different block/thread structure
    char visited [100]; // TODO figure out proper memory allocation technique within device to share accross all threads (should be citiesCount)
    int citySequence[100]; // TODO figure out proper memory allocation technique within device to share accross all threads (should be citiesCount)
    NextCity* nextCityProbabilities=0;


    int currentCity = ant;
    hiprandState localState = state[ant];
    for (int i = 0; i < citiesCount; i++) {
        visited[currentCity] = 1;
        citySequence[i] = currentCity;
        nextCityProbabilities = calculatePathsSelectionProbabilies(cityMap1D, fermoneMap1D, fermoneImportance, distanceImportance, currentCity, visited, citiesCount);
        double r = (double)hiprand_uniform(&localState);
        currentCity = selectNexyCity(nextCityProbabilities, r);
    }
    double distance = calculatePathDistance(citySequence, citiesCount, cityMap1D);


    distances[ant] = distance;
    
    //TODO transer for loop to parallel ??
    for (int i = 0; i < citiesCount; i++) {
        citySequences[ant * citiesCount + i] = citySequence[i];
    }
}

void evaporateFermone(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
    double* dev_fermone_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_fermone_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    evaporateFermoneKernel << <1, size >> > (dev_fermone_map, size, fermoneEvaporation);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(fermoneMap1D, dev_fermone_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    Error:
        hipFree(dev_fermone_map);
}

void scaleMap(double* scaledMap, const double* baseMap, double scaler, unsigned int size) {
    double* dev_scaled_map = 0;
    double* dev_base_map = 0;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_scaled_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_base_map, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_base_map, baseMap, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    scaleMapKernel << <1, size >> > (dev_scaled_map, dev_base_map, scaler, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(scaledMap, dev_scaled_map, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    Error:
        hipFree(dev_scaled_map);
        hipFree(dev_base_map);
}

void moveAnts(double* cityMap1D, double* distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences) {
    double* dev_city_map = 0;
    double* dev_distance_map = 0;
    double* dev_fermone_map = 0;
    double* dev_distances = 0;
    int* dev_city_sequences = 0;
    hipError_t cudaStatus;

    hiprandState* devStates=0;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_city_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distance_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_fermone_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distances, citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_city_sequences, mapSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&devStates, citiesCount * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_city_map, cityMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_distance_map, distanceMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }
    

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    setupKernel << <1, citiesCount >> > (devStates);

    moveAnt << <1, citiesCount >> > (dev_city_map, dev_distance_map, dev_fermone_map, mapSize, citiesCount, fermoneImportance, distanceImportance, dev_distances, dev_city_sequences, devStates);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(distances, dev_distances, citiesCount * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(citySequences, dev_city_sequences, mapSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


Error:
    hipFree(dev_city_map);
    hipFree(dev_fermone_map);
    hipFree(dev_distances);
    hipFree(dev_city_sequences);
}

void leaveFermones(double* fermoneMap1D, int* citySequences, double* distances, double fermoneIncrease, unsigned int mapSize, unsigned int citiesCount) {

    double* dev_fermone_map = 0;
    int* dev_city_sequences = 0;
    double* dev_distancecs = 0;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_fermone_map, mapSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_city_sequences, mapSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_distancecs, citiesCount * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_city_sequences, citySequences, mapSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_fermone_map, fermoneMap1D, mapSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_distancecs, distances, citiesCount * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


    leaveFermone << <citiesCount, citiesCount >> > (dev_fermone_map, dev_city_sequences, dev_distancecs, fermoneIncrease, citiesCount);


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(fermoneMap1D, dev_fermone_map, mapSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }


Error:
    hipFree(dev_fermone_map);
    hipFree(dev_city_sequences);
}

// Helper function for using CUDA to add vectors in parallel.
void addcuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

extern "C" {
    void add_ant_wpr(int* c, const int* a, const int* b, int size) {
        addcuda(c, a, b, size);
    }

    void scale_city_matrix_wrp(double* flatScaledCityMap, const double* flatCityMap, unsigned int size, double distanceScaler) {
        scaleMap(flatScaledCityMap, flatCityMap, distanceScaler, size);
    }

    void evaporate_fermone_wrp(double* fermoneMap1D, unsigned int size, double fermoneEvaporation) {
        evaporateFermone(fermoneMap1D, size, fermoneEvaporation);
    }


    void move_ants_wrp(double* cityMap1D, double * distanceMap1D, double* fermoneMap1D, unsigned int mapSize, unsigned int citiesCount, double fermoneImportance, double distanceImportance, double* distances, int* citySequences) {
        moveAnts(cityMap1D, distanceMap1D, fermoneMap1D, mapSize, citiesCount, fermoneImportance, distanceImportance, distances, citySequences);
    }

    void leave_fermone_wrp(double* fermoneMap1D, int* citySequences, double * distances, double fermoneIncrease, unsigned int mapSize, unsigned int cityCount) {
        leaveFermones(fermoneMap1D, citySequences, distances, fermoneIncrease, mapSize, cityCount);
    }
}
